#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>



// simple kernel function that adds two vectors
__global__ void vect_add(float *a, float *b, int N)
{
   int idx = threadIdx.x;
   if (idx<N) a[idx] = a[idx] + b[idx];
}

// function called from main fortran program
extern "C" void kernel_wrapper_(float *a, float *b, int *Np)
{
   float  *a_d, *b_d;  // declare GPU vector copies
   
   int blocks = 1;     // uses 1 block of
   int N = *Np;        // N threads on GPU

   // Allocate memory on GPU
   hipMalloc( (void **)&a_d, sizeof(float) * N );
   hipMalloc( (void **)&b_d, sizeof(float) * N );

   // copy vectors from CPU to GPU
   hipMemcpy( a_d, a, sizeof(float) * N, hipMemcpyHostToDevice );
   hipMemcpy( b_d, b, sizeof(float) * N, hipMemcpyHostToDevice );

   // call function on GPU
   vect_add<<< blocks, N >>>( a_d, b_d, N);

   // copy vectors back from GPU to CPU
   hipMemcpy( a, a_d, sizeof(float) * N, hipMemcpyDeviceToHost );
   hipMemcpy( b, b_d, sizeof(float) * N, hipMemcpyDeviceToHost );

   // free GPU memory
   hipFree(a_d);
   hipFree(b_d);
   return;
}
