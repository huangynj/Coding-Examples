
#include <hip/hip_runtime.h>
#include "stdio.h"

#define NUM_BLOCKS 32
#define NUM_THREADS_PER_BLOCK 32

// value is a pointer to a single integer
__global__ void testKernel(int* value) {
  if (blockIdx.x == 0 && threadIdx.x == 0) *value = 1;
}

int main(int argc, char** argv) {

  int value_cpu = 0; 

  int* value_gpu;

  hipMalloc((void**)&value_gpu,sizeof(float));
  hipMemcpy(value_gpu,&value_cpu,sizeof(float),hipMemcpyHostToDevice);
 
  /* Call the gpu kernel */
  testKernel<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(value_gpu);

  hipMemcpy(&value_cpu,value_gpu,sizeof(float),hipMemcpyDeviceToHost);
  if (value_cpu == 1) {
    printf("The cuda test passed GPU not corrupted\n");
    return 0;
  } else {
    printf("The cuda test failed the GPU is corrupted\n");
    return -1;
  }
}
